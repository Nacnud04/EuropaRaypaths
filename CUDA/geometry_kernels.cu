#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>
#include <math.h>

#include "elementary_kernels.cu"

__global__ void compIncidentRays(float sx, float sy, float sz,
                                 float* d_fx, float* d_fy, float* d_fz,
                                 float* d_fnx, float* d_fny, float* d_fnz,
                                 float* d_fux, float* d_fuy, float* d_fuz,
                                 float* d_fvx, float* d_fvy, float* d_fvz,
                                 float* d_Itd, float* d_Iph, float* d_Ith,
                                 int nfacets) {
    
    // first get the distance betweem the source and all facets
    pointDistanceBulk(sx, sy, sz,
                  d_fx, d_fy, d_fz,
                  d_Itd, nfacets);

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < nfacets) {

        // get incident cartesian vector
        float Ix = (d_fx[idx] - sx) / d_Itd[idx];
        float Iy = (d_fy[idx] - sy) / d_Itd[idx];
        float Iz = (d_fz[idx] - sz) / d_Itd[idx];

        // incident inclination
        d_Ith[idx] = slowArcCos(dotProduct(-1*Ix, -1*Iy, -1*Iz, 
                                          d_fnx[idx], d_fny[idx], d_fnz[idx]));

        // incident azimuth
        d_Iph[idx] = slowAtan2(
            dotProduct(Ix, Iy, Iz, d_fux[idx], d_fuy[idx], d_fuz[idx]),
            dotProduct(Ix, Iy, Iz, d_fvx[idx], d_fvy[idx], d_fvz[idx])
        );

    }    

}


__device__ hipFloatComplex facetReradiation(float dist, float th, float ph,  
                                  float lam, float fs){
    // NOTE: dist is the observer distance from the facet aperture

    // start with k
    float k = 2 * 3.14159265f / lam;

    // now make complex c = ((1j * r**2)/lam) * (np.exp(-1j*k*R)/R) * k 
    // (1j * r**2)/lam
    hipFloatComplex i_r2_over_lam = make_hipFloatComplex(0.0f, fs*fs / lam);
    // exp(-1j*k*R)/R
    float neg_kR = -1 * k * dist;
    hipFloatComplex exp_term = make_hipFloatComplex(cosf(neg_kR)/dist, sinf(neg_kR)/dist);
    // combine into c
    hipFloatComplex c_val = hipCmulf(i_r2_over_lam, exp_term);
    c_val = hipCmulf(c_val, make_hipFloatComplex(k, 0.0f));

    // now get sinc components:
    // np.sinc(((r) / lam) * np.sin(ph) * np.cos(th))
    float sinc1 = sinc((fs / lam) * slowSin(ph) * slowCos(th));
    // np.sinc(((r) / lam) * np.sin(ph) * np.sin(th))
    float sinc2 = sinc((fs / lam) * slowSin(ph) * slowSin(th));

    // combine all together
    return hipCmulf(c_val, make_hipFloatComplex(sinc1 * sinc2, 0.0f));

}


__device__ float radarEq(float P, float G, float sigma, float lam, float dist, float fs){
    
    // Pr = (Pt * G^2 * lam^2 * sigma) / ((4*pi)^3 * R^4)
    float num = P * G * G * lam * lam * sigma;
    float denom = pow(4 * 3.14159, 3) * dist * dist * dist * dist; // (4*pi)^3
    return (num / denom) * fs * fs;

}


__device__ float radarEqOneWay(float P, float G, float lam, float dist, float fs){
    
    // Pr = (Pt * G^2 * lam^2) / ((4*pi)^2 * R^2)
    float num = P * G * G * lam * lam;
    float denom = pow(4 * 3.14159, 2) * dist * dist; // (4*pi)^2
    return (num / denom) * fs * fs;

}


__global__ void compReflectedEnergy(float* d_Itd, float* d_Ith, float* d_Iph,
                                    hipFloatComplex* d_fRe, float* d_Rth, float* d_fRfrC,
                                    float P, float G, float sigma, float fs, float lam, 
                                    float nu1, float nu2, float alpha1, 
                                    float ks, int polarization, int nfacets){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < nfacets) {

        // first get facet reradiation
        d_fRe[idx] = facetReradiation(d_Itd[idx], d_Ith[idx], d_Iph[idx], lam, fs);

        // losses from radar equation
        d_fRe[idx] = hipCmulf(d_fRe[idx], make_hipFloatComplex(radarEq(P, G, sigma, lam, d_Itd[idx], fs), 0.0f));

        // reflection coefficient
        // horizontal pol.
        float rho_h;
        if (polarization == 0){ 
            rho_h = (nu2 * slowCos(d_Ith[idx]) - nu1 * slowCos(d_Rth[idx])) /
                    (nu2 * slowCos(d_Ith[idx]) + nu1 * slowCos(d_Rth[idx]));
        } 
        // vertical pol.
        else if (polarization == 1) {
            rho_h = (nu2 * slowCos(d_Rth[idx]) - nu1 * slowCos(d_Ith[idx])) /
                    (nu2 * slowCos(d_Rth[idx]) + nu1 * slowCos(d_Ith[idx]));
        }
        d_fRfrC[idx] = rho_h * rho_h;
        d_fRe[idx] = hipCmulf(d_fRe[idx], 
                             make_hipFloatComplex(1 - d_fRfrC[idx], 0.0f));

        // signal attenuation
        d_fRe[idx] = hipCmulf(d_fRe[idx], 
                             make_hipFloatComplex(expf(-2.0f * alpha1 * d_Itd[idx]), 0.0f));

        // surface roughness losses
        float rough_loss = expf(-4*((ks*slowCos(d_Ith[idx]))*(ks*slowCos(d_Ith[idx]))));
        d_fRe[idx] = hipCmulf(d_fRe[idx],
                             make_hipFloatComplex(rough_loss, 0.0f));

    }

}


__device__ float snellsLaw(float th, float eps_1, float eps_2) {

    float sin_th = slowSin(th);
    float sin_Rth = sqrtf(eps_1/eps_2) * sin_th;
    if (sin_Rth > 1.0f) {
        // total internal reflection set to something more than pi/2 so it is easy to identify
        return 1e4;
    } else {
        return slowArcSin(sin_Rth);
    }

}


__global__ void compRefractedRays(float* d_Ith, float* d_Iph,
                                  float* d_Rth, float* d_Rtd,
                                  float* d_fx, float* d_fy, float* d_fz,
                                  float tx, float ty, float tz,
                                  float eps_1, float eps_2, int nfacets) {

    pointDistanceBulk(tx, ty, tz,
                      d_fx, d_fy, d_fz,
                      d_Rtd, nfacets);

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < nfacets) {

        // refracted inclination
        d_Rth[idx] = snellsLaw(d_Ith[idx], eps_1, eps_2);

    }    

}


__global__ void compRefrEnergyIn(
                    float* d_Itd, float* d_Iph,
                    float* d_Rtd, float* d_Rth, float* d_fRfrC,
                    hipFloatComplex* d_fRefrEI, float* d_fRfrSR,
                    float ks, int nfacets, float alpha2, float c1, float c2,
                    float fs, float P, float G, float lam) {

    float c = 299792458.0f; // speed of light in m/s

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < nfacets) {

        // start with facet reradiation
        d_fRefrEI[id] = facetReradiation(d_Rtd[id], d_Rth[id], d_Iph[id], lam, fs);

        // losses from radar equation
        d_fRefrEI[id] = hipCmulf(d_fRefrEI[id], make_hipFloatComplex(radarEqOneWay(P, G, lam, d_Itd[id]+d_Rtd[id], fs), 0.0f));

        // refraction coefficient
        d_fRefrEI[id] = hipCmulf(d_fRefrEI[id], 
                               make_hipFloatComplex(d_fRfrC[id], 0.0f));

        // signal attenuation
        d_fRefrEI[id] = hipCmulf(d_fRefrEI[id], 
                               make_hipFloatComplex(expf(-2 * alpha2 * d_Rtd[id]), 0.0f));

        // surface roughness losses
        float rough_loss = expf(-4*((ks*slowCos(d_Rth[id]))*(ks*slowCos(d_Rth[id]))));
        d_fRefrEI[id] = hipCmulf(d_fRefrEI[id],
                                 make_hipFloatComplex(rough_loss, 0.0f));

        // total travel slant range
        d_fRfrSR[id] = d_Itd[id] * (c / c1) +  d_Rtd[id] * (c / c2);

    }

}


__global__ void compRefrEnergyOut(float* d_Itd, float* d_Iph,
                                  float* d_Rtd, float* d_Rth,
                                  hipFloatComplex* d_fRefrEO, float* d_fRfrC, 
                                  float ks, int nfacets, float alpha1, float alpha2, float c1, float c2,
                                  float fs, float G, float lam){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < nfacets) {

        // first get facet reradiation
        d_fRefrEO[idx] = facetReradiation(d_Itd[idx], d_Rth[idx], -1*d_Iph[idx], lam, fs);

        // losses from radar equation
        // we dont need power here as this acts to scale the power from the inbound path
        d_fRefrEO[idx] = hipCmulf(d_fRefrEO[idx], 
                         make_hipFloatComplex(radarEqOneWay(1, G, lam, d_Rtd[idx]+d_Itd[idx], fs), 0.0f));

        // for transmission coefficient use refraction cofficient from before
        // THIS MIGHT NOT BE RIGHT
        d_fRefrEO[idx] = hipCmulf(d_fRefrEO[idx], 
                             make_hipFloatComplex(d_fRfrC[idx], 0.0f));

        // signal attenuation
        // first above surface
        d_fRefrEO[idx] = hipCmulf(d_fRefrEO[idx], 
                             make_hipFloatComplex(expf(-2.0f * alpha1 * d_Itd[idx]), 0.0f));
        // then in subsurface
        d_fRefrEO[idx] = hipCmulf(d_fRefrEO[idx], 
                             make_hipFloatComplex(expf(-2.0f * alpha2 * d_Rtd[idx]), 0.0f));

        // surface roughness losses
        float rough_loss = expf(-4*((ks*slowCos(d_Rth[idx]))*(ks*slowCos(d_Rth[idx]))));
        d_fRefrEO[idx] = hipCmulf(d_fRefrEO[idx],
                             make_hipFloatComplex(rough_loss, 0.0f));

    }

}