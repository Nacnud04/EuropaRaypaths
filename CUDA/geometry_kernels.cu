#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>
#include <math.h>

// thrust includes for aperture cropping
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/functional.h>

#include "elementary_kernels.cu"

const float pi = 3.14159265f;

// function which approximates the amount of illuminated facets within the aperture
// assuming the facets are arranged in a grid
__host__ int nIlluminatedFacets(float sz, float fz, float fs, float theta) {
    
    // get radius of illuminated area on surface
    float r;
    r = (sz - fz) * sinf((pi/180)*theta);

    // turn into area
    float A;
    A = pi * r * r;

    // get the amount of illuminated facets
    float nfacets;
    nfacets = A / (fs * fs);

    // add a buffer amount
    float buff = 4.0f;

    return nfacets * buff;

}

__global__ void compIncidentRays(float sx, float sy, float sz,
                                 float* d_fx, float* d_fy, float* d_fz,
                                 float* d_fnx, float* d_fny, float* d_fnz,
                                 float* d_fux, float* d_fuy, float* d_fuz,
                                 float* d_fvx, float* d_fvy, float* d_fvz,
                                 float* d_Itd, float* d_Iph, float* d_Ith,
                                 int nfacets) {
    
    // first get the distance betweem the source and all facets
    pointDistanceBulk(sx, sy, sz,
                  d_fx, d_fy, d_fz,
                  d_Itd, nfacets);

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < nfacets) {

        // get incident cartesian vector
        float Ix = (d_fx[idx] - sx) / d_Itd[idx];
        float Iy = (d_fy[idx] - sy) / d_Itd[idx];
        float Iz = (d_fz[idx] - sz) / d_Itd[idx];

        // incident inclination
        d_Ith[idx] = slowArcCos(dotProduct(-1*Ix, -1*Iy, -1*Iz, 
                                          d_fnx[idx], d_fny[idx], d_fnz[idx]));

        // incident azimuth
        d_Iph[idx] = slowAtan2(
            dotProduct(Ix, Iy, Iz, d_fux[idx], d_fuy[idx], d_fuz[idx]),
            dotProduct(Ix, Iy, Iz, d_fvx[idx], d_fvy[idx], d_fvz[idx])
        );

    }    

}


struct is_below_threshold
{
    const float thresh;
    __host__ __device__
    is_below_threshold(float t) : thresh(t) {}

    __host__ __device__
    bool operator()(const float x) const {
        return x < thresh;
    }
};

int cropByAperture(int totfacets, int nfacets, float aperture,
                    float* d_Ffx,  float* d_Ffy,  float* d_Ffz,
                    float* d_Ffnx, float* d_Ffny, float* d_Ffnz,
                    float* d_Ffux, float* d_Ffuy, float* d_Ffuz,
                    float* d_Ffvx, float* d_Ffvy, float* d_Ffvz,
                    float* d_FItd, float* d_FIph, float* d_FIth,
                    float* d_fx,  float* d_fy,  float* d_fz,
                    float* d_fnx, float* d_fny, float* d_fnz,
                    float* d_fux, float* d_fuy, float* d_fuz,
                    float* d_fvx, float* d_fvy, float* d_fvz,
                    float* d_Itd, float* d_Iph, float* d_Ith){

    // wrap raw pointers with the thrust device pointer
    thrust::device_ptr<float> Ffx(d_Ffx),     Ffy(d_Ffy),   Ffz(d_Ffz);
    thrust::device_ptr<float> Ffnx(d_Ffnx), Ffny(d_Ffny), Ffnz(d_Ffnz);
    thrust::device_ptr<float> Ffux(d_Ffux), Ffuy(d_Ffuy), Ffuz(d_Ffuz);
    thrust::device_ptr<float> Ffvx(d_Ffvx), Ffvy(d_Ffvy), Ffvz(d_Ffvz);
    thrust::device_ptr<float> FItd(d_FItd), FIph(d_FIph), FIth(d_FIth);

    // define output thrust device pointers (point at the destination/cropped arrays)
    thrust::device_ptr<float> fx(d_fx),     fy(d_fy),   fz(d_fz);
    thrust::device_ptr<float> fnx(d_fnx), fny(d_fny), fnz(d_fnz);
    thrust::device_ptr<float> fux(d_fux), fuy(d_fuy), fuz(d_fuz);
    thrust::device_ptr<float> fvx(d_fvx), fvy(d_fvy), fvz(d_fvz);
    thrust::device_ptr<float> Itd(d_Itd), Iph(d_Iph), Ith(d_Ith);

    // define predicate
    is_below_threshold pred((pi/180)*aperture);

    // copy_if using d_FIth as a stencil
    auto end_fx = thrust::copy_if(Ffx, Ffx + totfacets, FIth, fx, pred);
    auto end_fy = thrust::copy_if(Ffy, Ffy + totfacets, FIth, fy, pred);
    auto end_fz = thrust::copy_if(Ffz, Ffz + totfacets, FIth, fz, pred);

    auto end_fnx = thrust::copy_if(Ffnx, Ffnx + totfacets, FIth, fnx, pred);
    auto end_fny = thrust::copy_if(Ffny, Ffny + totfacets, FIth, fny, pred);
    auto end_fnz = thrust::copy_if(Ffnz, Ffnz + totfacets, FIth, fnz, pred);

    auto end_fux = thrust::copy_if(Ffux, Ffux + totfacets, FIth, fux, pred);
    auto end_fuy = thrust::copy_if(Ffuy, Ffuy + totfacets, FIth, fuy, pred);
    auto end_fuz = thrust::copy_if(Ffuz, Ffuz + totfacets, FIth, fuz, pred);

    auto end_fvx = thrust::copy_if(Ffvx, Ffvx + totfacets, FIth, fvx, pred);
    auto end_fvy = thrust::copy_if(Ffvy, Ffvy + totfacets, FIth, fvy, pred);
    auto end_fvz = thrust::copy_if(Ffvz, Ffvz + totfacets, FIth, fvz, pred);

    auto end_Itd = thrust::copy_if(FItd, FItd + totfacets, FIth, Itd, pred);
    auto end_Iph = thrust::copy_if(FIph, FIph + totfacets, FIth, Iph, pred);
    auto end_Ith = thrust::copy_if(FIth, FIth + totfacets, FIth, Ith, pred);

    int valid = (int)(end_fx - fx);
    return valid;
}

__global__ void compTargetRays(float tx, float ty, float tz,
                               float* d_fx, float* d_fy, float* d_fz,
                               float* d_fnx, float* d_fny, float* d_fnz,
                               float* d_fux, float* d_fuy, float* d_fuz,
                               float* d_fvx, float* d_fvy, float* d_fvz,
                               float* d_Ttd, float* d_Tph, float* d_Tth,
                               int nfacets) {
    
    // first get the distance betweem the source and all facets
    pointDistanceBulk(tx, ty, tz,
                      d_fx, d_fy, d_fz,
                      d_Ttd, nfacets);

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < nfacets) {

        // get incident cartesian vector
        float Ix = (d_fx[idx] - tx) / d_Ttd[idx];
        float Iy = (d_fy[idx] - ty) / d_Ttd[idx];
        float Iz = (d_fz[idx] - tz) / d_Ttd[idx];

        // incident inclination
        d_Tth[idx] = slowArcCos(dotProduct(-1*Ix, -1*Iy, -1*Iz, 
                                          d_fnx[idx], d_fny[idx], d_fnz[idx]));

        // incident azimuth
        d_Tph[idx] = slowAtan2(
            dotProduct(Ix, Iy, Iz, d_fux[idx], d_fuy[idx], d_fuz[idx]),
            dotProduct(Ix, Iy, Iz, d_fvx[idx], d_fvy[idx], d_fvz[idx])
        );

    }    

}


__device__ float facetReradiation(float dist, float th, float ph,  
                                  float lam, float fs){
    // NOTE: dist is the observer distance from the facet aperture

    // start with k
    float k = 2 * 3.14159265f / lam;

    // now make complex c = ((1j * r**2)/lam) * (np.exp(-1j*k*R)/R) * k 
    // (1j * r**2)/lam
    hipFloatComplex i_r2_over_lam = make_hipFloatComplex(0.0f, fs*fs / lam);
    // exp(-1j*k*R)/R
    float neg_kR = -1 * k * dist;
    hipFloatComplex exp_term = make_hipFloatComplex(cosf(neg_kR)/dist, sinf(neg_kR)/dist);
    // combine into c
    hipFloatComplex c_val = hipCmulf(i_r2_over_lam, exp_term);
    c_val = hipCmulf(c_val, make_hipFloatComplex(k, 0.0f));

    // now get sinc components:
    // np.sinc(((r) / lam) * np.sin(ph) * np.cos(th))
    float sinc1 = sinc((fs / lam) * sinGPU(th) * cosGPU(ph));
    // np.sinc(((r) / lam) * np.sin(ph) * np.sin(th))
    float sinc2 = sinc((fs / lam) * sinGPU(th) * sinGPU(ph));

    // combine all together
    return hipCabsf(hipCmulf(c_val, make_hipFloatComplex(sinc1 * sinc2, 0.0f)));

}


__device__ float radarEq(float P, float G, float sigma, float lam, float dist, float fs){
    
    // Pr = (Pt * G^2 * lam^2 * sigma) / ((4*pi)^3 * R^4)
    float num = P * G * G * lam * lam * sigma;
    float denom = pow(4 * 3.14159, 3) * dist * dist * dist * dist; // (4*pi)^3
    return (num / denom) * fs * fs;

}


__global__ void compReflectedEnergy(float* d_Itd, float* d_Ith, float* d_Iph,
                                    float* d_fRe, float* d_Rth, float* d_fRfrC,
                                    float P, float G, float sigma, float fs, float lam, 
                                    float nu1, float nu2, float alpha1, 
                                    float ks, int polarization, int nfacets){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < nfacets) {

        // first get facet reradiation
        // we double inclination angle to as the center of the beam pattern is in the
        // exact opposite direction as the incident ray. 
        d_fRe[idx] = facetReradiation(d_Itd[idx], 2*d_Ith[idx], -1*d_Iph[idx], lam, fs);

        // losses from radar equation
        d_fRe[idx] = d_fRe[idx] * radarEq(P, G, sigma, lam, d_Itd[idx], fs);

        // reflection coefficient
        // horizontal pol.
        float rho;
        if (polarization == 0){ 
            rho = (nu2 * cosGPU(d_Ith[idx]) - nu1 * cosGPU(d_Rth[idx])) /
                    (nu2 * cosGPU(d_Ith[idx]) + nu1 * cosGPU(d_Rth[idx]));
            d_fRfrC[idx] = 1 - (rho * rho);
        } 
        // vertical pol.
        else if (polarization == 1) {
            rho = (nu2 * cosGPU(d_Rth[idx]) - nu1 * cosGPU(d_Ith[idx])) /
                    (nu2 * cosGPU(d_Rth[idx]) + nu1 * cosGPU(d_Ith[idx]));
            d_fRfrC[idx] = 1 - (rho * rho);
        }
        d_fRe[idx] = d_fRe[idx] * rho * rho;

        // signal attenuation
        d_fRe[idx] = d_fRe[idx] * expf(-2.0f * alpha1 * d_Itd[idx]);

        // surface roughness losses
        float rough_loss = expf(-4*((ks*cosGPU(d_Ith[idx]))*(ks*cosGPU(d_Ith[idx]))));
        d_fRe[idx] = d_fRe[idx] * rough_loss;

    }

}


__device__ float snellsLaw(float th, float eps_1, float eps_2) {

    float sin_th = sinGPU(th);
    float sin_Rth = sqrtf(eps_1/eps_2) * sin_th;
    if (sin_Rth > 1.0f) {
        // total internal reflection set to something more than pi/2 so it is easy to identify
        return 1e4;
    } else {
        return slowArcSin(sin_Rth);
    }

}


__global__ void compRefractedRays(float* d_Ith, float* d_Iph,
                                  float* d_Rth, float* d_Rtd,
                                  float* d_fx, float* d_fy, float* d_fz,
                                  float tx, float ty, float tz,
                                  float eps_1, float eps_2, int nfacets) {

    pointDistanceBulk(tx, ty, tz,
                      d_fx, d_fy, d_fz,
                      d_Rtd, nfacets);

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < nfacets) {

        // refracted inclination
        d_Rth[idx] = snellsLaw(d_Ith[idx], eps_1, eps_2);

    }    

}


__global__ void compRefrEnergyIn(
                    float* d_Rtd, float* d_Rth, float* d_Itd, float* d_Iph,
                    float* d_Ttd, float* d_Tth, float* d_Tph, float* d_fRfrC,
                    float* d_fRefrEI, float* d_fRfrSR,
                    float ks, int nfacets, float alpha2, float c1, float c2,
                    float fs, float P, float G, float lam) {

    float c = 299792458.0f; // speed of light in m/s

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < nfacets) {

        // start with facet reradiation
        // we need to get delta theta between refracted and forced ray as follows:
        float delta_th = (pi - d_Rth[id]) - d_Tth[id];
        // now we do similar for phi
        float delta_ph = d_Iph[id] - d_Tph[id];
        // compute facet reradiation
        d_fRefrEI[id] = facetReradiation(d_Ttd[id], delta_th, delta_ph, lam, fs);

        // refraction coefficient
        d_fRefrEI[id] = d_fRefrEI[id] * d_fRfrC[id];

        // signal attenuation
        d_fRefrEI[id] = d_fRefrEI[id] * expf(-2 * alpha2 * d_Rtd[id]);

        // surface roughness losses
        float rough_loss = expf(-4*((ks*cosGPU(d_Rth[id]))*(ks*cosGPU(d_Rth[id]))));
        d_fRefrEI[id] = d_fRefrEI[id] * rough_loss;

        // total travel slant range
        d_fRfrSR[id] = d_Itd[id] + d_Rtd[id];

    }

}


__global__ void compRefrEnergyOut(float* d_Itd, float* d_Iph,
                                  float* d_Ttd, float* d_Tth, float* d_Tph,
                                  float* d_fRefrEO, float* d_fRfrC, 
                                  float ks, int nfacets, float alpha1, float alpha2, float c1, float c2,
                                  float fs, float G, float lam, float eps_1, float eps_2){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < nfacets) {

        // for facet reradiation for upward transmitted, we can assume that
        // RrTh is the upward transmitted inclination angle
        float RrTh = snellsLaw(pi - d_Tth[idx], eps_2, eps_1);

        d_fRefrEO[idx] = facetReradiation(d_Itd[idx], RrTh, d_Tph[idx]-d_Iph[idx], lam, fs);

        // for transmission coefficient use refraction cofficient from before
        d_fRefrEO[idx] = d_fRefrEO[idx] * d_fRfrC[idx];

        // signal attenuation
        // first above surface
        d_fRefrEO[idx] = d_fRefrEO[idx] * expf(-2.0f * alpha1 * d_Itd[idx]);
        // then in subsurface
        d_fRefrEO[idx] = d_fRefrEO[idx] * expf(-2.0f * alpha2 * d_Ttd[idx]);

        // surface roughness losses
        float rough_loss = expf(-4*((ks*cosGPU(d_Tth[idx]))*(ks*cosGPU(d_Tth[idx]))));
        d_fRefrEO[idx] = d_fRefrEO[idx] * rough_loss;

    }

}